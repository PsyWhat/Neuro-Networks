#include "hip/hip_runtime.h"


#pragma once


#include "CUDAFunctions.h"








#define BLOCK_SIZE 32







__global__ void matBlockMultKernel( Matrix a , Matrix b , Matrix res )
{
	int blockRow = blockIdx.y;
	int blockColumn = blockIdx.x;


	double Cvalue = 0.0;

	int row = threadIdx.y;
	int col = threadIdx.x;

	int rowm = row + blockRow * blockDim.y;
	int colm = col + blockColumn * blockDim.x;

	int addition = (a.colums % BLOCK_SIZE == 0) ? 0 : 1;

	for ( int m = 0; m < (a.colums / BLOCK_SIZE) + addition; ++m )
	{

		__shared__ double As[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];


		int rown = (m * blockDim.y + row);
		int coln = (m * blockDim.x + col);

		if ( rowm < a.rows &&  coln < a.colums )
		{
			As[row][col] = a.values[rowm + a.rows * coln];
		} else
		{
			As[row][col] = 0.0;
		}

		if ( rown < b.rows && colm < b.colums )
		{
			Bs[row][col] = b.values[rown + b.rows * colm];
		} else
		{
			Bs[row][col] = 0.0;
		}

		__syncthreads();

		for ( int e = 0; e < BLOCK_SIZE; ++e )
		{
			Cvalue += As[row][e] * Bs[e][col];
		}

		__syncthreads();

	}
	if ( rowm < res.rows && colm < res.colums )
	{
		res.values[rowm + res.rows * colm] = Cvalue;
	}
}


__host__
Matrix* CudaMatMult( Matrix *A , Matrix *B , int device )
{
	Matrix *res = NULL;

	Matrix a , b , r;
	hipError_t err;
	int deviceCount;

	a.rows = A->rows;
	a.colums = A->colums;
	a.values = NULL;

	b.rows = B->rows;
	b.colums = B->colums;
	b.values = NULL;

	r.rows = A->rows;
	r.colums = B->colums;
	r.values = NULL;


	if ( hipGetDeviceCount( &deviceCount ) < device )
	{
		err = hipError_t::hipErrorInvalidDevice;
		goto FREE;
	}

	err = hipSetDevice( device );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMalloc( (void**)(&(a.values)) , sizeof( double ) * a.colums * a.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMalloc( (void**)(&b.values) , sizeof( double ) * b.colums * b.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMalloc( (void**)(&r.values) , sizeof( double ) * r.colums * r.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

	err = hipMemcpy( (void*)a.values , A->values , sizeof( double ) * a.colums * a.rows , hipMemcpyHostToDevice );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMemcpy( b.values , B->values , sizeof( double ) * b.colums * b.rows , hipMemcpyHostToDevice );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

	dim3 blockSize( BLOCK_SIZE , BLOCK_SIZE );
	dim3 numBlocks( (r.colums / BLOCK_SIZE) + ((r.colums % BLOCK_SIZE == 0) ? 0 : 1) , (r.rows / BLOCK_SIZE) + ((r.rows % BLOCK_SIZE == 0) ? 0 : 1) );

	matBlockMultKernel << <numBlocks , blockSize >> > (a , b , r);

	err = hipPeekAtLastError();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipGetLastError();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipDeviceSynchronize();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	//checkCudaErrors( hipPeekAtLastError() );
	//checkCudaErrors( hipDeviceSynchronize() );


	res = (Matrix*)malloc( sizeof( Matrix ) );
	res->colums = r.colums;
	res->rows = r.rows;
	res->values = NULL;
	res->values = (double*)malloc( sizeof( double ) * res->colums * res->rows );

	err = hipMemcpy( res->values , r.values , sizeof( double ) * (res->colums) * (res->rows) , hipMemcpyDeviceToHost );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

FREE:

	hipFree( a.values );
	hipFree( b.values );
	hipFree( r.values );

	if ( err != hipSuccess && res != NULL )
	{
		if ( res->values != NULL )
		{
			free( res->values );
		}
		free( res );
		res = NULL;
	}


	return res;
}





__global__
void CudaAddMatsKernell( Matrix a , Matrix b , Matrix r )
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	if ( row < r.rows && col < r.colums )
	{
		r.values[row + col * r.rows] = a.values[row + col * a.rows] + b.values[row + col * b.rows];
	}

}



__host__
Matrix* CudaAddMats( Matrix *A , Matrix *B , int device /*= 0 */ )
{
	Matrix *res = NULL;

	Matrix a , b , r;
	hipError_t err;
	int deviceCount;

	a.rows = A->rows;
	a.colums = A->colums;
	a.values = NULL;

	b.rows = B->rows;
	b.colums = B->colums;
	b.values = NULL;

	r.rows = A->rows;
	r.colums = A->colums;
	r.values = NULL;


	if ( hipGetDeviceCount( &deviceCount ) < device )
	{
		err = hipError_t::hipErrorInvalidDevice;
		goto FREE;
	}

	err = hipSetDevice( device );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMalloc( (void**)(&(a.values)) , sizeof( double ) * a.colums * a.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMalloc( (void**)(&b.values) , sizeof( double ) * b.colums * b.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMalloc( (void**)(&r.values) , sizeof( double ) * r.colums * r.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

	err = hipMemcpy( (void*)a.values , A->values , sizeof( double ) * a.colums * a.rows , hipMemcpyHostToDevice );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMemcpy( b.values , B->values , sizeof( double ) * b.colums * b.rows , hipMemcpyHostToDevice );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

	dim3 blockSize( BLOCK_SIZE , BLOCK_SIZE );
	dim3 numBlocks( (r.colums / BLOCK_SIZE) + ((r.colums % BLOCK_SIZE == 0) ? 0 : 1) , (r.rows / BLOCK_SIZE) + ((r.rows % BLOCK_SIZE == 0) ? 0 : 1) );

	CudaAddMatsKernell << <numBlocks , blockSize >> > (a , b , r);

	err = hipPeekAtLastError();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipGetLastError();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipDeviceSynchronize();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	//checkCudaErrors( hipPeekAtLastError() );
	//checkCudaErrors( hipDeviceSynchronize() );


	res = (Matrix*)malloc( sizeof( Matrix ) );
	res->colums = r.colums;
	res->rows = r.rows;
	res->values = NULL;
	res->values = (double*)malloc( sizeof( double ) * res->colums * res->rows );

	err = hipMemcpy( res->values , r.values , sizeof( double ) * (res->colums) * (res->rows) , hipMemcpyDeviceToHost );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

FREE:

	hipFree( a.values );
	hipFree( b.values );
	hipFree( r.values );

	if ( err != hipSuccess && res != NULL )
	{
		if ( res->values != NULL )
		{
			free( res->values );
		}
		free( res );
		res = NULL;
	}


	return res;
}







__global__
void CudaHadamardKernell( Matrix a , Matrix b , Matrix r )
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	if ( row < r.rows && col < r.colums )
	{
		r.values[row + col * r.rows] = a.values[row + col * a.rows] * b.values[row + col * b.rows];
	}

}


__host__
Matrix* CudaHadamardProduct( Matrix *A , Matrix *B , int device /*= 0 */ )
{

	Matrix *res = NULL;

	Matrix a , b , r;
	hipError_t err;
	int deviceCount;

	a.rows = A->rows;
	a.colums = A->colums;
	a.values = NULL;

	b.rows = B->rows;
	b.colums = B->colums;
	b.values = NULL;

	r.rows = A->rows;
	r.colums = A->colums;
	r.values = NULL;


	if ( hipGetDeviceCount( &deviceCount ) < device )
	{
		err = hipError_t::hipErrorInvalidDevice;
		goto FREE;
	}

	err = hipSetDevice( device );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMalloc( (void**)(&(a.values)) , sizeof( double ) * a.colums * a.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMalloc( (void**)(&b.values) , sizeof( double ) * b.colums * b.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMalloc( (void**)(&r.values) , sizeof( double ) * r.colums * r.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

	err = hipMemcpy( (void*)a.values , A->values , sizeof( double ) * a.colums * a.rows , hipMemcpyHostToDevice );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMemcpy( b.values , B->values , sizeof( double ) * b.colums * b.rows , hipMemcpyHostToDevice );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

	dim3 blockSize( BLOCK_SIZE , BLOCK_SIZE );
	dim3 numBlocks( (r.colums / BLOCK_SIZE) + ((r.colums % BLOCK_SIZE == 0) ? 0 : 1) , (r.rows / BLOCK_SIZE) + ((r.rows % BLOCK_SIZE == 0) ? 0 : 1) );

	CudaHadamardKernell << <numBlocks , blockSize >> > (a , b , r);

	err = hipPeekAtLastError();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipGetLastError();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipDeviceSynchronize();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	//checkCudaErrors( hipPeekAtLastError() );
	//checkCudaErrors( hipDeviceSynchronize() );


	res = (Matrix*)malloc( sizeof( Matrix ) );
	res->colums = r.colums;
	res->rows = r.rows;
	res->values = NULL;
	res->values = (double*)malloc( sizeof( double ) * res->colums * res->rows );

	err = hipMemcpy( res->values , r.values , sizeof( double ) * (res->colums) * (res->rows) , hipMemcpyDeviceToHost );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

FREE:

	hipFree( a.values );
	hipFree( b.values );
	hipFree( r.values );

	if ( err != hipSuccess && res != NULL )
	{
		if ( res->values != NULL )
		{
			free( res->values );
		}
		free( res );
		res = NULL;
	}


	return res;
}


__global__
void CudaMultDoubleKernell( Matrix a , double d , Matrix r )
{
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	int col = threadIdx.x + blockDim.x * blockIdx.x;

	if ( row < r.rows && col < r.colums )
	{
		r.values[row + col * r.rows] = a.values[row + col * a.rows] * d;
	}
}

__host__
Matrix* CudaMultDouble( Matrix *A , double d, int device )
{
	Matrix *res = NULL;

	Matrix a  , r;
	hipError_t err;
	int deviceCount;

	a.rows = A->rows;
	a.colums = A->colums;
	a.values = NULL;


	r.rows = A->rows;
	r.colums = A->colums;
	r.values = NULL;


	if ( hipGetDeviceCount( &deviceCount ) < device )
	{
		err = hipError_t::hipErrorInvalidDevice;
		goto FREE;
	}

	err = hipSetDevice( device );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipMalloc( (void**)(&(a.values)) , sizeof( double ) * a.colums * a.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	
	err = hipMalloc( (void**)(&r.values) , sizeof( double ) * r.colums * r.rows );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

	err = hipMemcpy( (void*)a.values , A->values , sizeof( double ) * a.colums * a.rows , hipMemcpyHostToDevice );
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	

	dim3 blockSize( BLOCK_SIZE , BLOCK_SIZE );
	dim3 numBlocks( (r.colums / BLOCK_SIZE) + ((r.colums % BLOCK_SIZE == 0) ? 0 : 1) , (r.rows / BLOCK_SIZE) + ((r.rows % BLOCK_SIZE == 0) ? 0 : 1) );

	CudaMultDoubleKernell <<<numBlocks , blockSize >> > (a , d , r);

	err = hipPeekAtLastError();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipGetLastError();
	if ( err != hipError_t::hipSuccess )
		goto FREE;
	err = hipDeviceSynchronize();
	if ( err != hipError_t::hipSuccess )
		goto FREE;

	res = (Matrix*)malloc( sizeof( Matrix ) );
	res->colums = r.colums;
	res->rows = r.rows;
	res->values = NULL;
	res->values = (double*)malloc( sizeof( double ) * res->colums * res->rows );

	err = hipMemcpy( res->values , r.values , sizeof( double ) * (res->colums) * (res->rows) , hipMemcpyDeviceToHost );
	if ( err != hipError_t::hipSuccess )
		goto FREE;

FREE:

	hipFree( a.values );
	hipFree( r.values );

	if ( err != hipSuccess && res != NULL )
	{
		if ( res->values != NULL )
		{
			free( res->values );
		}
		free( res );
		res = NULL;
	}


	return res;
}




